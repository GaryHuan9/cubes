#include "Rendering/Engine.hpp"
#include "Rendering/Packets.hpp"
#include "Rendering/Kernels.hpp"
#include "Rendering/Accumulator.hpp"
#include "Scenic/Camera.hpp"

namespace cb
{

class KernelLaunch
{
public:
	explicit KernelLaunch(unsigned int region, unsigned int block_size = 256)
	{
		block_count = dim3((region + block_size - 1) / block_size);
		this->block_size = dim3(block_size);
	}

	explicit KernelLaunch(const UInt2& region, unsigned int block_size = 16)
	{
		UInt2 count = (region + UInt2(block_size - 1)) / block_size;
		block_count = dim3(count.x(), count.y());
		this->block_size = dim3(block_size, block_size);
	}

	explicit KernelLaunch(const UInt3& region, unsigned int block_size = 8)
	{
		UInt3 count = (region + UInt3(block_size - 1)) / block_size;
		block_count = dim3(count.x(), count.y(), count.z());
		this->block_size = dim3(block_size, block_size, block_size);
	}

	template<typename Kernel, typename... Arguments>
	void launch(const Kernel& kernel, const Arguments& ... arguments)
	{
		kernel<<<block_count, block_size>>>(arguments...);
	}

private:
	dim3 block_count;
	dim3 block_size;
};

static constexpr size_t Capacity = 1024 * 1024;
static Camera* device_camera;

Engine::Engine()
{
	new_path_packets = CudaVector<NewPathPackets>(Capacity);
	trace_packets = CudaVector<TracePackets>(Capacity);
	hit_packets = CudaVector<HitPacket>(Capacity);

	Camera camera;

	camera.set_position(Float3(0.0f, 0.0f, -3.0f));

	cuda_check(hipMalloc(&device_camera, sizeof(Camera)));
	cuda_copy(device_camera, &camera);
}

Engine::~Engine() = default;

void Engine::change_resolution(const UInt2& new_resolution)
{
	if (resolution == new_resolution) return;
	resolution = new_resolution;

	uint32_t count = resolution.x() * resolution.y();
	accumulators = CudaArray<Accumulator>(count, true);
}

void Engine::render()
{
	new_path_packets.clear();
	trace_packets.clear();
	hit_packets.clear();

	uint32_t count = resolution.x() * resolution.y();
	accumulators = CudaArray<Accumulator>(count, true);

	KernelLaunch(Capacity).launch(kernels::render_begin, resolution, new_path_packets.view(new_path_packets.capacity()));
	KernelLaunch(Capacity).launch(kernels::new_path, resolution, device_camera, new_path_packets, trace_packets);
	KernelLaunch(Capacity).launch(kernels::trace_rays, trace_packets, hit_packets.view(trace_packets.size()));
	KernelLaunch(Capacity).launch(kernels::render_end, resolution, trace_packets.view(), hit_packets.view(), accumulators);
}

void Engine::output(hipSurfaceObject_t surface_object) const
{
	//	KernelLaunch(resolution).launch(kernels::test, resolution, accumulators, 0.0f);
	KernelLaunch(resolution).launch(kernels::output, resolution, accumulators, surface_object);
}

} // cb
