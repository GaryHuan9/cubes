#include "Utilities/CudaUtilities.hpp"
#include "Utilities/Vector.hpp"

#include <stdexcept>

namespace cb
{

__host__
void cuda_check(hipError_t error)
{
	if (error == hipError_t::hipSuccess) return;
	std::string error_string(hipGetErrorString(error));
	throw std::runtime_error("CUDA error: " + error_string);
}

__device__
uint32_t get_thread_index()
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

}
