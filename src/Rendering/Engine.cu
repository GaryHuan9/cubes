#include "Rendering/Engine.hpp"
#include "Rendering/Structures.hpp"
#include "Rendering/Kernels.hpp"
#include "Scenic/Camera.hpp"

namespace cb
{

class KernelLaunch
{
public:
	explicit KernelLaunch(unsigned int region, unsigned int block_size = 256)
	{
		block_count = dim3((region + block_size - 1) / block_size);
		this->block_size = dim3(block_size);
	}

	explicit KernelLaunch(const UInt2& region, unsigned int block_size = 16)
	{
		UInt2 count = (region + UInt2(block_size - 1)) / block_size;
		block_count = dim3(count.x(), count.y());
		this->block_size = dim3(block_size, block_size);
	}

	explicit KernelLaunch(const UInt3& region, unsigned int block_size = 8)
	{
		UInt3 count = (region + UInt3(block_size - 1)) / block_size;
		block_count = dim3(count.x(), count.y(), count.z());
		this->block_size = dim3(block_size, block_size, block_size);
	}

	template<typename Kernel, typename... Arguments>
	void launch(const Kernel& kernel, const Arguments& ... arguments)
	{
		kernel<<<block_count, block_size>>>(arguments...);
	}

private:
	dim3 block_count;
	dim3 block_size;
};

Engine::Engine()
{
	cuda_check(hipMalloc(&camera, sizeof(Camera)));

	paths = CudaArray<Path>(Capacity);
	randoms = CudaArray<hiprandState>(Capacity);
	KernelLaunch(Capacity).launch(kernels::new_random, randoms);

	trace_queries = CudaVector<TraceQuery>(Capacity);
	material_queries = CudaVector<MaterialQuery>(Capacity);
	escape_packets = CudaVector<EscapedPacket>(Capacity);
}

Engine::~Engine() = default;

void Engine::change_resolution(const UInt2& new_resolution)
{
	if (resolution == new_resolution) return;
	resolution = new_resolution;

	uint32_t count = resolution.x() * resolution.y();
	accumulators = CudaArray<Accumulator>(count);
	reset_render();
}

void Engine::change_camera(const Camera& new_camera)
{
	cuda_copy(camera, &new_camera);
	reset_render();
}

void Engine::reset_render()
{
	accumulators.clear();
	index_start = resolution.x() * resolution.y() / 2;
	index_start -= Capacity / 2;
}

void Engine::render()
{
	hipEvent_t start_event, end_event;
	hipEventCreate(&start_event);
	hipEventCreate(&end_event);
	hipEventRecord(start_event);

	cuda_check(hipDeviceSynchronize());

	trace_queries.clear();
	material_queries.clear();
	escape_packets.clear();

	KernelLaunch launcher(Capacity);

	index_start %= resolution.x() * resolution.y();
	size_t start = index_start;
	index_start += Capacity;

	launcher.launch(kernels::new_path, paths, resolution, start, camera, trace_queries);

	for (size_t depth = 0; depth < 16; ++depth)
	{
		launcher.launch(kernels::trace, trace_queries);
		launcher.launch(kernels::shade, trace_queries, material_queries, escape_packets, randoms);

		cuda_check(hipDeviceSynchronize());
		trace_queries.clear();

		launcher.launch(kernels::diffuse, material_queries);
		launcher.launch(kernels::advance, material_queries, trace_queries, paths);
		launcher.launch(kernels::escaped, escape_packets, paths);

		cuda_check(hipDeviceSynchronize());
		material_queries.clear();
		escape_packets.clear();
	}

	launcher.launch(kernels::accumulate, paths, start, accumulators);

	hipEventRecord(end_event);
	hipEventSynchronize(end_event);

	//	float milliseconds;
	//	cuda_check(hipEventElapsedTime(&milliseconds, start_event, end_event));
	//	std::printf("Sampling took %f ms\n", milliseconds);
}

void Engine::output(hipSurfaceObject_t surface_object) const
{
	KernelLaunch(resolution).launch(kernels::output, resolution, accumulators, surface_object);
}

}
